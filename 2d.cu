#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 5
#define M 6
__global__ void dkernel(unsigned *matrix){
  i = threadIdx.x
  j = threadIdx.y

  matrix[threadIdx.x*blockDim.y+threadIdx.y] = threadIdx.x*blockDim.y+threadIdx.y 
  
}
int main() {
  dim3 block(N,M,1);
  unsigned *gpu_matrix, *device_matrix;
  hipMalloc(&gpu_matrix, N*M*sizeof(unsigned));
  device_matrix = (unsigned *) malloc(N*M *sizeof(unsigned)) 
  dkernel<<<1,block>>>(device_matrix);
  hipMemcpy(device_matrix, gpu_matrix, N*M*sizeof(unsigned), hipMemcpyDeviceToHost);
  for(unsigned i =0; i<N;i++)
  {
    for(int j=0;j<M;j++)
    {
      printf("%2d", device_matrix[i*M+j]);
    }
    printf("\n");
  }
  hipDeviceSynchronize();
  return 0;
}