#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

struct node
{
  struct node *next;
  int data;
};

struct node *createNode(int data)
{
  struct node *newNode = (struct node *) malloc (sizeof(struct node));
  newNode->next= null;
  newNode->data = data;
}

struct node *createList()
{
  struct node *head = NULL;

  for(int i=20; i >= 0; --i)
  {
    struct node *newNode = createNode(i);
    newNode->next = head;
    head = newNode;
  }
  return head;
}

_device__ __host__ void printList(struct node *head){
  if(head)
  {
    printf("%d\t", head->data);
    printList(head->next);
  }
  else
    printf("\n");
}

struct node *copyNode(struct node *newNode)
{
  struct node newNodeGPU;
  hipMalloc(&newNodeGPU, sizeof(struct node));
  hipMemcpy(newNodeGPU, newNode,sizeof(struct node), cudaMemxpyHostToDevice)
  return newNodeGPU; 
} 

struct node *copyList(struct node *head)
{
  if(!head) return NULL;
  struct node newNode;
  newNode.next = copyList(head->next);
  newNode.data = head->data;
  return copyNode(&newNode); 
} 

__global__ void printListGPU(struct node *head) {
  printList(head);
}
#define BLOCKSIZE 1024
int main() {
 unsigned N = 1024;
 struct node *head = createList();
 struct node *gpuhead = copyList(head);
 hipDeviceSynchronize();
 printList(head);
 printListGPU<<<1,1 >>> (gpuhead);
 hipDeviceSynchronize();

 return 0;
}
