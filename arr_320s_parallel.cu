#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void init(int *a, int alen){
  unsigned id = threadIdx.x;
  if(id<alen)
  {
    a[id] = 0;
  }
}
__global__ void dkernel(int *a, int alen){
  unsigned id = threadIdx.x;
  if(id<alen)
  {
    a[id] += id;
  }
}


int main()
{
  int *da, N;
  N=1024;
  int a[N];
  hipMalloc(&da,sizeof(int)*N);
  init<<<1,N>>>(da,N);
  dkernel<<<1,N>>>(da,N);
  hipMemcpy(a,da,N*sizeof(int),hipMemcpyDeviceToHost);
  for (int i=0;i<N;i++)
    printf("%d\t",a[i]);
  // cudaDeviceSynchronize();
  return 0;
}