#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 100
__global__ void dkernel(int *a){
    a[threadIdx.x] = threadIdx.x*threadIdx.x;
}

int main()
{
  // da is GPU memory pointer
  int a[N], *da;
  hipMalloc(&da, sizeof(int)*N);
  dkernel<<<1,N>>>(da);
  // copy memory from GPU (Device) to CPU (host) - dest, src
  hipMemcpy(a,da,N*sizeof(int),hipMemcpyDeviceToHost);
  for (int i=0;i<N;i++)
    printf("%d\t",a[i]);
  // cudaDeviceSynchronize();
  return 0;
}