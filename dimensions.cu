#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void dkernel(){
   if (threadIdx.x ==0 && blockIdx.x ==0 &&
   threadIdx.y ==0 && blockIdx.y ==0 &&
   threadIdx.z ==0 && blockIdx.z ==0)
   {
    printf("%d %d %d %d %d %d\n", gridDim.x,gridDim.y,gridDim.z, blockDim.x, blockDim.y, blockDim.z);
   }
}
int main() {
  dim3 grid(2,3,4); //blockIds
  dim3 block(5,6,7); //threadIds
  dkernel<<<grid,block>>>();
  hipDeviceSynchronize();
  return 0;
}