#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void dkernel(int *a, int *sq, int *cu, int alen){
  unsigned id = threadIdx.x;
  if(id<alen)
  {
    // printf("%d ", sq[id]);    printf("%d ", cu[id]);

    a[id] = (sq[id]*sq[id]) + (cu[id]*cu[id]*cu[id]);
  }
}


int main()
{

  FILE *cubed = fopen("ints.txt", "r");  
  FILE *squared = fopen("another_ints.txt", "r"); 
  if (cubed == NULL || squared == NULL) {
        printf("Unable to open the file\n");
        return 1;
    } 
    int sq, cu, i;
    int N = 10;
    int sq_arr[N], cu_arr[N],a[N];
    while ((fscanf(cubed, "%d", &cu) == 1) && (fscanf(squared, "%d", &sq) == 1)) {
        sq_arr[i] = sq;
        cu_arr[i] = cu;
        i+=1;
    }
  fclose(squared);
  fclose(cubed);
    
    int *da, *dsq_arr, *dcu_arr;
    hipMalloc(&da, sizeof(int)*N);
    hipMalloc(&dsq_arr, sizeof(int)*N);
    hipMalloc(&dcu_arr, sizeof(int)*N);

    hipMemcpy(dcu_arr,cu_arr,N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dsq_arr,sq_arr,N*sizeof(int), hipMemcpyHostToDevice);

    dkernel<<<1, N>>>(da,dsq_arr, dcu_arr, N);

    hipMemcpy(a,da,N*sizeof(int), hipMemcpyDeviceToHost);
  
    for(int i=0;i<N;i++)
    {
      printf("%d ", a[i]);
    }


  return 0;
}