#include <stdio.h>
#include <hip/hip_runtime.h>
__global__ void dkernel(unsigned *vector, unsigned vectorsize) {
 unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
 if (id < vectorsize) vector[id] = id;
}
#define BLOCKSIZE 1024
int main(int nn, char *str[]) {
 unsigned N = 2000;
 unsigned *vector, *hvector;
 hipMalloc(&vector, N * sizeof(unsigned));
 hvector = (unsigned *)malloc(N * sizeof(unsigned));
 unsigned nblocks = ceil((float)N / BLOCKSIZE);
 printf("nblocks = %d\n", nblocks);
 dkernel<<<nblocks, BLOCKSIZE>>>(vector, N);
 hipMemcpy(hvector, vector, N * sizeof(unsigned), hipMemcpyDeviceToHost);
 for (unsigned ii = 0; ii < N; ++ii) {
 printf("%4d ", hvector[ii]);
 }
 return 0;
}