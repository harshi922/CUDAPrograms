#include <stdio.h>
#include <hip/hip_runtime.h>
__global__ void dkernel(unsigned *matrix, unsigned *res, unsigned size) {
  unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
  for(unsigned jj=0;jj<size;jj++)
  {
    for(unsigned kk=0;kk<size;kk++)
    {
      res[id * size + jj] += matrix[id * size + kk] * matrix[kk * size + jj];
    }
    // res[id * size + jj] = sum;
  }
}
#define N 3
int main() {
 unsigned *matrix, *hmatrix, *res, *hres;
 hipMalloc(&matrix, N * N *sizeof(unsigned));
 hipMalloc(&res, N * N *sizeof(unsigned));

 hmatrix = (unsigned *)malloc(N * N * sizeof(unsigned));
 hres = (unsigned *)malloc(N * N * sizeof(unsigned));

 for (unsigned ii = 0; ii < N; ++ii) {
    for(unsigned jj = 0; jj < N; ++jj)
    {
      hmatrix[ii *N + jj] = ii *N + jj;
      printf("%4d ", hmatrix[ii *N + jj]);
    }
 }  for (unsigned ii = 0; ii < N; ++ii) {
    for(unsigned jj = 0; jj < N; ++jj)
    {
      hres[ii *N + jj] = 0;
    }
 } 


 hipMemcpy(matrix, hmatrix, N * N * sizeof(unsigned), hipMemcpyHostToDevice);
 dkernel<<<1, N>>>(matrix, res, N);
 hipMemcpy(hres, res, N * N * sizeof(unsigned), hipMemcpyDeviceToHost);
 for (unsigned ii = 0; ii < N; ++ii) {
    for(unsigned jj = 0; jj < N; ++jj)
    {
      printf("%4d ", hres[ii *N + jj]);
    }
 }
 return 0;
}