#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 100
__global__ void dkernel(){
    printf("%d\n",threadIdx.x*threadIdx.x);
}

int main()
{
  dkernel<<<1,N>>>();
  hipDeviceSynchronize();
  return 0;
}