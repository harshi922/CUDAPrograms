#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 5
#define M 6
__global__ void dkernel(unsigned *matrix){

  matrix[threadIdx.x+blockDim.x*blockIdx.x] = threadIdx.x+blockDim.x*blockIdx.x; 
  
}
int main() {
  unsigned *gpu_matrix, *device_matrix;
  hipMalloc(&gpu_matrix, N*M*sizeof(unsigned));
  device_matrix = (unsigned *) malloc(N*M *sizeof(unsigned)); 
  dkernel<<<N,M>>>(gpu_matrix);
  hipMemcpy(device_matrix, gpu_matrix, N*M*sizeof(unsigned), hipMemcpyDeviceToHost);
  for(unsigned i =0; i<N;i++)
  {
    for(int j=0;j<M;j++)
    {
      printf("%2d", device_matrix[i*M+j]);
    }
    printf("\n");
  }
  hipDeviceSynchronize();
  return 0;
}