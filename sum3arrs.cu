#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>  // Include for timing

// Error checking macro
#define CUDA_ERROR_CHECK(call) {                       \
    hipError_t err = call;                           \
    if (err != hipSuccess) {                         \
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err)); \
        exit(1);                                      \
    }                                                 \
}
#define COMPARE_ARRAYS(h_a, h_b, h_c, h_res, size) {       \
    for (int i = 0; i < size; i++) {                  \
        if (h_res[i] != h_a[i] + h_b[i] + h_c[i]) {            \
            printf("Mismatch at index %d: %d + %d + %d != %d\n", \
                   i, h_a[i], h_b[i], h_c[i], h_res[i]);     \
            exit(1);                                   \
        }                                             \
    }                                                 \
    printf("Arrays match!\n");                         \
}

__global__ void dkernel(int *a, int *b, int *c, int *res, int size)
{
    int gid = threadIdx.x + blockDim.x * blockIdx.x;
    if(gid < size)
    {
        res[gid] = a[gid] + b[gid] + c[gid];
    } 
}

void addarrs(int *a, int *b, int *c, int *res, int size)
{
    for (int i = 0; i < size; i++)
    {
        res[i] = a[i] + b[i] + c[i];
    }
}

int main()
{
    int size = 1 << 22;  // Set size to 2^22
    int *h_a, *h_b, *h_c, *h_res, *d_a, *d_b, *d_c, *d_res;

    // Allocate host memory
    h_a = (int*)malloc(size * sizeof(int));
    h_b = (int*)malloc(size * sizeof(int));
    h_c = (int*)malloc(size * sizeof(int));
    h_res = (int*)malloc(size * sizeof(int));

    // Initialize host arrays
    for (int i = 0; i < size; i++) 
    {
        h_a[i] = rand() % 100;
        h_b[i] = rand() % 100;
        h_c[i] = rand() % 100;
    }
    memset(h_res, 0, size * sizeof(int));

    // Allocate device memory
    CUDA_ERROR_CHECK(hipMalloc((void **)&d_a, size * sizeof(int)));
    CUDA_ERROR_CHECK(hipMalloc((void **)&d_b, size * sizeof(int)));
    CUDA_ERROR_CHECK(hipMalloc((void **)&d_c, size * sizeof(int)));
    CUDA_ERROR_CHECK(hipMalloc((void **)&d_res, size * sizeof(int)));

    // Copy data from host to device
    CUDA_ERROR_CHECK(hipMemcpy(d_a, h_a, size * sizeof(int), hipMemcpyHostToDevice));
    CUDA_ERROR_CHECK(hipMemcpy(d_b, h_b, size * sizeof(int), hipMemcpyHostToDevice));
    CUDA_ERROR_CHECK(hipMemcpy(d_c, h_c, size * sizeof(int), hipMemcpyHostToDevice));

    // Define block sizes to test
    int block_sizes[] = { 64, 128, 256, 512 };
    
    // Loop over different block sizes
    for(int i = 0; i < 4; i++)
    {
        int block_size = block_sizes[i];
        int grid_size = (size + block_size - 1) / block_size; // Calculate grid size

        // Timing GPU execution
        clock_t start_gpu = clock(); // Start timing
        dkernel<<<grid_size, block_size>>>(d_a, d_b, d_c, d_res, size);
        CUDA_ERROR_CHECK(hipDeviceSynchronize()); // Wait for GPU to finish
        clock_t end_gpu = clock(); // End timing
        double gpu_time = (double)(end_gpu - start_gpu) / CLOCKS_PER_SEC; // Calculate elapsed time

        // Copy result from device to host
        CUDA_ERROR_CHECK(hipMemcpy(h_res, d_res, size * sizeof(int), hipMemcpyDeviceToHost));

        // Timing CPU execution
        clock_t start_cpu = clock(); // Start timing
        addarrs(h_a, h_b, h_c, h_res, size);
        clock_t end_cpu = clock(); // End timing
        double cpu_time = (double)(end_cpu - start_cpu) / CLOCKS_PER_SEC; // Calculate elapsed time

        // Compare CPU and GPU results
        COMPARE_ARRAYS(h_a, h_b, h_c, h_res, size);

        // Print the timings
        printf("Block size: %d\n", block_size);
        printf("GPU execution time: %f seconds\n", gpu_time);
        printf("CPU execution time: %f seconds\n", cpu_time);
        printf("\n"); // Add a new line for better readability
    }

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_res);
    
    // Free host memory
    free(h_a);
    free(h_b);
    free(h_c);
    free(h_res);

    return 0;
}
