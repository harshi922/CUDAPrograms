#include <stdio.h>
#include <hip/hip_runtime.h>
__global__ void idxing(int *arr, int size)
{
  int gid = blockIdx.z * gridDim.x * gridDim.y * blockDim.x*blockDim.y*blockDim.z 
  + blockIdx.y * gridDim.x *  blockDim.x*blockDim.y*blockDim.z
  + blockIdx.x * blockDim.x*blockDim.y*blockDim.z
  + threadIdx.z * blockDim.x * blockDim.y
  + threadIdx.y * blockDim.x
  + threadIdx.x;
  if (gid<size)
  {
    printf("%d %d\n", gid,arr[gid]);
  } 

}

int main()
{
  int *h_arr,*d_arr;
  int size=64;
  h_arr = (int*)malloc(sizeof(int)*size);
  for(int i=0;i<size;i++)
  {
    h_arr[i] = i*100;
  }
  hipMalloc(&d_arr,sizeof(int)*size);
  hipMemcpy(d_arr,h_arr,sizeof(int)*size,hipMemcpyHostToDevice);
  dim3 grid(4,4,4);
  dim3 block(2,2,2);
  idxing<<<grid, block>>>(d_arr,size);
  hipDeviceSynchronize();
  return 0;
}